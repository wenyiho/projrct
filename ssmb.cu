#define M_PI 3.14159265358979323846


#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <ctgmath>
#include <ctime>
//#include <complex>

// For the CUDA runtime routines (prefixed with "cuda_")
//#include <cuda.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand.h>
#include <hip/hip_complex.h>
//#include <cuda_runtime.h>
//#include <device_launch_parameters.h>

#define NUM_OF_THREADS 10000
#define THREADS_PER_BLOCK 256

#define TURNS   25000										        	/*number of revolution*/
#define NE      10000 										        	/*number of electron*/

#define NUMOFZPT 300								                	/*number of segmentation of phase*/
//double numofzptdb = numofzpt;

//#define _C_m_s		(2.99792458e8)										/*c-m/s*/
//#define R56			(0.0512e-3) 										/*dispersion-m*/
#define ES			(0.44e-3)											/*energy spread*/
#define DAMPRATE	(1.45e-4)    										/*damping rate*/
//#define EBE			(0.629e9)				   							/*beam energy-GeV*/
//#define MANONOR 	(0.3e6)												/*modulation amplitude-GeV*/

#define MA			(0.00047694753577106518) 							/*normalized modulation amplitude*/
//double my_gamma = EBE/(0.511e6);				      					/*Lorentz factor*/

#define NUMMB		(20.0)    								    		/*number of microbunch*/

#define QNEP		(7.4929300010076163e-006)  							/*for quantum excitation*/

#define MODWL		(1.0e-6)  						 		    		/*modulation wavelength-m*/
#define NORR56		(321.69908772759482)				    			/*normalized dispersion*/

#define DN			(1.3646097851959425e-005)

#define LPRANGE		(125.66370614359172)								/*phase range in longitudinal phase space*/

//double zposegdb;

__constant__ int seeding = 1;

__device__ double atomicDoubleAdd(double* address, double val){
    unsigned long long int* address_as_ull =
                             (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}

__device__ hipDoubleComplex ft(double *x){
	hipDoubleComplex value = make_hipDoubleComplex(0.0, 0.0);//, I = make_cuDoubleComplex(0.0, 1.0);
	int l;
	double y = -2.0*M_PI/(double)NUMOFZPT*20.0;
	hipDoubleComplex tmp;
	for(l = 0; l < NUMOFZPT; l++){
		sincos(y*(double)l, &tmp.x, &tmp.y);
		
		value.x += x[l]*tmp.x;
		value.y += x[l]*tmp.y;
		//printf("%f\n",x[l]);
		x[l] = 0;
	}
	return value;
}

// Kernel
__global__ void CalKernel(double *lp, double *lpth, double *bf, hiprandStateXORWOW_t *states){

	int tid = threadIdx.x, id = tid + blockIdx.x * THREADS_PER_BLOCK;
	hiprandStateXORWOW_t localState = states[id];
	__shared__ double zdis[NUMOFZPT];
	
	double l_lp, l_lpth;
	hipDoubleComplex l_bf;
	
	int zposeg;
                                                                            /*energy array*/
    double iniu;															/*for Gaussian random number*/
    double iniv;															/*for Gaussian random number*/
	double err;
	if(id < NE){
		iniu = hiprand_uniform_double(&localState);
		iniv = hiprand_uniform_double(&localState);
		l_lp = hiprand_uniform_double(&localState)*LPRANGE;
		l_lpth = sqrt(-2.0*log(iniu))*cos(2.0*M_PI*iniv)*ES;
		//if(id == 0) printf("\nlp[id] = %f, lpth[id] = %f\n",lp[id],lpth[id]);
		int i;

		if(id == 0){
			for(i = 0; i < NUMOFZPT; i++){
				zdis[i] = 0;
			}
		}
		for(i = 0; i < TURNS; i++){
			__syncthreads();
			//if(id == 0 && i < 10) printf("\n**TURN: %d**\n",i);
			l_lp += MA*sin(l_lp);

			iniu = hiprand_uniform_double(&localState);
			iniv = hiprand_uniform_double(&localState);
			err = -DAMPRATE*l_lpth + QNEP*sqrt(-2.0*log(iniu))*cos(2.0*M_PI*iniv);
			//if(id == 0 && i < 10) printf("err = %f\n",err);
			iniu = hiprand_uniform_double(&localState);
			iniv = hiprand_uniform_double(&localState);
			l_lp += NORR56*l_lpth+ (err + DN*sqrt(-2.0*log(iniu))*cos(2.0*M_PI*iniv))*NORR56/2.0;
			l_lpth = l_lpth + err;
			//if(id == 0 && i < 10) printf("lp[id] = %f, lpth[id] = %f\n",lp[id],lpth[id]);

			l_lp = fmod(l_lp,LPRANGE);
			zposeg = (l_lp/(LPRANGE/(double)NUMOFZPT));
			
			if(zposeg >= 0) {atomicDoubleAdd(&zdis[zposeg], 1.0);}
			
			__syncthreads();
			if(threadIdx.x == 0){
				l_bf = ft(zdis);
				atomicDoubleAdd(&bf[i], l_bf.x);
				atomicDoubleAdd(&bf[i+TURNS], l_bf.y);
			}
		}
		if(l_lp < 0.0) l_lp = l_lp + LPRANGE;
		
		lp[id] = l_lp;
		lpth[id] = l_lpth;
	}
}

__global__ void SetupKernel(hiprandStateXORWOW_t *states){
    int id = threadIdx.x + blockIdx.x * THREADS_PER_BLOCK;
	if(id < NE){
		hiprand_init(seeding, id, 0, &states[id]);
	}
}

void Calculate(double *lp, double *lpth, double *bf, int blocksPerGrid, hiprandStateXORWOW_t *states){
    hipError_t error;
	double *d_lp, *d_lpth, *d_bf, bf_tmp[TURNS*2];
	
	//cudaMemcpyToSymbol(blockcounter, &counter, sizeof(int));

	// Allocate memory for result on Device
    hipMalloc(&d_lp, sizeof(double)*NE);
    hipMalloc(&d_lpth, sizeof(double)*NE);
    hipMalloc(&d_bf, sizeof(double)*TURNS*2);
	hipMemset(d_bf, 0, sizeof(double)*TURNS*2);

	// Launch Kernel
    CalKernel<<<blocksPerGrid, THREADS_PER_BLOCK>>>(d_lp, d_lpth, d_bf, states);
  // check for error
  error = hipGetLastError();
  if(error != hipSuccess)
  {
    // print the CUDA error message and exit
    printf("CUDA error: %s\n", hipGetErrorString(error));
    exit(-1);
  }

	// Copy result to Host
    error = hipMemcpy(lp, d_lp, sizeof(double)*NE, hipMemcpyDeviceToHost);
	//printf("Error NO:%d\n", error);
	printf("CUDA error: %s\n", hipGetErrorString(error));
    hipMemcpy(lpth, d_lpth, sizeof(double)*NE, hipMemcpyDeviceToHost);
    hipMemcpy(bf_tmp, d_bf, sizeof(double)*TURNS*2, hipMemcpyDeviceToHost);
	
	int i;
	for(i = 0; i < TURNS; i++){
		//if(i < 10) printf("(%f, %f i)\n",bf_tmp[i],bf_tmp[i+TURNS]);
		bf[i] = (bf_tmp[i]/NE)*(bf_tmp[i]/NE) + (bf_tmp[i+TURNS]/NE)*(bf_tmp[i+TURNS]/NE);
		bf[i] = sqrt(bf[i]);
	}

	// Free Memory
    hipFree(d_lp);
    hipFree(d_lpth);
    hipFree(d_bf);
}

/*void SetupConstant(){
	// Calculate constant value
	double l_ma = MANONOR/EBE;
	double l_qnep = ES*sqrt(2.0*DAMPRATE);
	double l_norr56 = 2.0*M_PI*R56/MODWL;
	double l_dn = 1.0/sqrt(M_PI*my_gamma/137.0)*DAMPRATE/2.0;
	double l_lprange = 2.0*M_PI*NUMMB;
	
	printf("\nma = %f\nqnep = %f\nnorr56 = %f\ndn = %f\nlprange = %f\n",l_ma,l_qnep,l_norr56,l_dn,l_lprange);
	
	// Copy constant value to device
	cudaMemcpyToSymbol(ma, &l_ma, sizeof(double));
	cudaMemcpyToSymbol(qnep, &l_qnep, sizeof(double));
	cudaMemcpyToSymbol(norr56, &l_norr56, sizeof(double));
	cudaMemcpyToSymbol(dn, &l_dn, sizeof(double));
	cudaMemcpyToSymbol(lprange, &l_lprange, sizeof(double));
}*/

void CalOnDevice(double *lp, double *lpth, double *bf){
    int blocksPerGrid = (NUM_OF_THREADS + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    hiprandStateXORWOW_t *states;

	// Allocate memory for Random Generator State
    hipMalloc((void **)&states, THREADS_PER_BLOCK * blocksPerGrid * sizeof(hiprandStateXORWOW_t));

	// Setup Constant
	/*printf("Setup Constant...");
	SetupConstant();
	printf("Complete.\n");*/
	
	// Setup Random Generator State
	printf("Setup Random Generator State...");
    SetupKernel<<<blocksPerGrid, THREADS_PER_BLOCK>>>(states);
	printf("Complete.\n");
	
	// Start Calculation
	printf("Start Calculation...");
    Calculate(lp, lpth, bf, blocksPerGrid, states);
	printf("Complete.\n");
	
    hipFree(states);
}

int main() {
	FILE *fpout;
	fpout = fopen("out.txt","w");
	FILE *fpoutt;
	fpoutt = fopen("outt.txt","w");
	FILE *fpouttt;
	fpouttt = fopen("outtt.txt","w");

	double lp[NE];															/*phase array*/
	double lpth[NE];
	double bf[TURNS];
	//double segpb = ((double)numofzpt)/nummb;

	printf("Execute calculation on the device.\n");
	CalOnDevice(lp, lpth, bf);

	int j;

	printf("Output: out.txt\n");
	for(j = 0; j < TURNS; j++){
        fprintf(fpout,"%f\n",sqrt(bf[j]));
	}
	printf("Output: outt.txt\n");
	for(j = 0; j < NE; j++){
        fprintf(fpoutt,"%f\n",lp[j]);
	}
	printf("Output: outtt.txt\n");
	for(j = 0; j < NE; j++){
        fprintf(fpouttt,"%f\n",lpth[j]);
	}

	fclose(fpout);
	fclose(fpoutt);
	fclose(fpouttt);

	return 0;
}
